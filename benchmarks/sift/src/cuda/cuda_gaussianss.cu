#include "hip/hip_runtime.h"
/********************************
Author: Sravanthi Kota Venkata
********************************/

#include "sift.h"

F2D* resizeArray1(F2D* array, int omin)
{
    F2D* prev = NULL;
    F2D* current = array;
    int o;
    if(omin<0)
    {
        for(o=1; o>=-omin; o--)
        {
            prev = current;
            current = doubleSize(current);
            fFreeHandle(prev);
        }
    }
    if(omin>0)
    {
        for(o=1; o<= omin; o++)
        {
            prev = current;
            current = halveSize(current);
            fFreeHandle(prev);
        }
    }
    return current;
}

/**
    Returns the Gaussian scale space of image I. Image I is assumed to be
    pre-smoothed at level SIGMAN. O,S,OMIN,SMIN,SMAX,SIGMA0 are the
    parameters of the scale space.
**/

F2D** cuda_gaussianss(F2D* array, float sigman, int O, int S, int omin, int smin, int smax, float sigma0)
{
   /* We compute the following items in the function
    1. Smooth input image per octave
    2. Smooth each octave for different intervals
    3. Subtract each "interval-1" smooth image from "interval" image per octave. So, per octave, we have "interval" * DOG images.
    4. So, octave * intervals * image
    5. Note: At each octave, the image is scaled down in both x and y directions
    */

    float k, dsigma0, dsigma;
    int s, i, j, o, so, M, N, sbest;
    int intervals = smax-smin+1;
    float temp, target_sigma, prev_sigma;
    F2D *TMP, **gss;
    F2D* I = array;

    // Scale multiplicative step
    k = pow(2, (1.0/S));
    dsigma0 = sigma0 * sqrt(1-(1.0/pow(k,2)));  // Scale step factor

    // If omin < 0, multiply the size of the image.
    I = resizeArray1(I, omin);
    M = I->height;
    N = I->width;
    so = -smin+1;       // Index offset

    gss = (F2D**) malloc(O*intervals*sizeof(F2D*));
    if(gss == NULL)
    {
        printf("Could not allocate memory\n");
        return NULL;
    }

    temp = sqrt(pow((sigma0*pow(k,smin)),2) - pow((sigman/pow(2,omin)),2));

    {
        gss[0] = fSetArray(I->height, I->width, 0);
        cuda_imsmooth(I, temp, gss[0] );

    }

    for(s=smin; s<smax; s++)
    {
        dsigma = pow(k,s+1) * dsigma0;
        gss[s+so] = fSetArray(gss[s+so-1]->height, gss[s+so-1]->width, 0);
        cuda_imsmooth( gss[(s+so-1)] , dsigma, gss[(s+so)] );
    }

    for(o=1; o<O; o++)
    {
        sbest = MIN(smin+S-1, smax-1);
        TMP = halveSize( gss[(o-1)*intervals+sbest+so]);
        target_sigma = sigma0 * pow(k,smin);
        prev_sigma = sigma0 * pow(k, (sbest+1)-S);

        temp = sqrt(pow(target_sigma,2) - pow(prev_sigma, 2));
        if(target_sigma > prev_sigma)
        {
            gss[o*intervals] = fSetArray(TMP->height, TMP->width, 0);
            cuda_imsmooth(TMP, temp, gss[o*intervals] );

        }
        else
        {
            int i;
            gss[o*intervals] = fSetArray(TMP->height, TMP->width, 0);
            for(i=0; i<(TMP->height*TMP->width); i++)
                asubsref(gss[o*intervals],i) = asubsref(TMP,i);
        }

        M = TMP->height;
        N = TMP->width;

        fFreeHandle(TMP);

        for(s=smin; s<smax; s++)
        {
            // The other levels are determined as above for the first octave.
            dsigma = pow(k,s+1) * dsigma0;
            gss[o*intervals+s+so] = fSetArray(gss[o*intervals+s-1+so]->height, gss[o*intervals+s-1+so]->width, 0);
            cuda_imsmooth( gss[o*intervals+s-1+so] , dsigma, gss[o*intervals+s+so]);
        }
    }

    fFreeHandle(I);

    return gss;
}

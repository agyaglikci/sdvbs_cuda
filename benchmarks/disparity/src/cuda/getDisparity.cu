/********************************
Author: Sravanthi Kota Venkata
********************************/

#include <stdio.h>
#include <stdlib.h>
#include "disparity.h"
//#include "cuda_disparity.cu"

/*
#define GPUERRCHK { gpuAssert((hipGetLastError()), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
*/

I2D* getDisparity(I2D* Ileft, I2D* Iright, int win_sz, int max_shift, int use_gpu, int gpu_transfer)
{
    I2D* retDisp;
    int nr, nc, k;
    I2D *halfWin;
    int half_win_sz, rows, cols;
    F2D *retSAD, *minSAD, *SAD, *integralImg;
    I2D* IrightPadded, *IleftPadded, *Iright_moved;
    // device-side copies
    I2D* d_retDisp;
    F2D *d_retSAD, *d_minSAD, *d_SAD, *d_integralImg;
    I2D* d_IrightPadded, *d_IleftPadded, *d_Iright_moved;
    
    nr = Ileft->height;
    nc = Ileft->width;
    half_win_sz=win_sz/2;
    
    
    minSAD = fSetArray(nr, nc, 255.0*255.0);
    retDisp = iSetArray(nr, nc,max_shift);
    halfWin = iSetArray(1,2,half_win_sz);

        if(win_sz > 1)
        {
            IleftPadded = padarray2(Ileft, halfWin);
            IrightPadded = padarray2(Iright, halfWin);
        }
        else
        {
            IleftPadded = Ileft;
            IrightPadded = Iright;
        }
    
    rows = IleftPadded->height;
    cols = IleftPadded->width;
    SAD = fSetArray(rows, cols,255);
    integralImg = fSetArray(rows, cols,0);
    retSAD = fMallocHandle(rows-win_sz, cols-win_sz);
    Iright_moved = iSetArray(rows, cols, 0);

    if(gpu_transfer) 
    {
      unsigned int* start_transfer = cudaStartTransfer();
      printf("doing gpu transfer\n");
      d_IleftPadded = iMallocAndCopy(IleftPadded);
      d_IrightPadded = iMallocAndCopy(IrightPadded);
      d_Iright_moved = iMallocAndCopy(Iright_moved);
      d_SAD = fMallocAndCopy(SAD);
      d_integralImg = fMallocAndCopy(integralImg);
      d_retSAD = fMallocAndCopy(retSAD);
      d_minSAD = fMallocAndCopy(minSAD);
      d_retDisp = iMallocAndCopy(retDisp);
      GPUERRCHK;
      cudaEndTransfer(start_transfer);
    }
    
    for( k=0; k<max_shift; k++)
    {    
        if(use_gpu) 
        {
          cuda_correlateSAD_2D(d_IleftPadded, d_IrightPadded, d_Iright_moved, win_sz, k, nr, nc, SAD, integralImg, retSAD);
          cuda_findDisparity(d_retSAD, d_minSAD, d_retDisp, k, nr, nc);
          GPUERRCHK;
        }
        else
        {
          correlateSAD_2D(IleftPadded, IrightPadded, Iright_moved, win_sz, k, SAD, integralImg, retSAD);
          findDisparity(retSAD, minSAD, retDisp, k, nr, nc);
        }
    }
    //printf("done with kernel\n");
    
    fFreeHandle(retSAD);
    fFreeHandle(minSAD);
    fFreeHandle(SAD);
    fFreeHandle(integralImg);
    iFreeHandle(halfWin);
    iFreeHandle(IrightPadded);
    iFreeHandle(IleftPadded);
    iFreeHandle(Iright_moved);
     
    if(gpu_transfer) 
    {
      unsigned int* start_transfer = cudaStartTransfer();
      hipFree(d_retSAD);
      hipFree(d_minSAD);
      hipFree(d_SAD);
      hipFree(d_integralImg);
      hipFree(d_IrightPadded);
      hipFree(d_IleftPadded);
      hipFree(d_Iright_moved);
      if(use_gpu) {
        iCopyFromGPU(retDisp, d_retDisp);
      }
      GPUERRCHK;
      cudaEndTransfer(start_transfer);
    }
    return retDisp;
}

